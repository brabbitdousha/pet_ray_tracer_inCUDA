#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <time.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include ""
#include "vec3.h"

#define checkCudaErrors(val) check_cuda((val),#val,_FILE_,_LINE_)

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error =" << static_cast<unsigned int>(result) << "at" <<
            file << ":" << line << "'" << func << "'\n'";

        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(vec3* fb, int max_x, int max_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    fb[pixel_index] = vec3(float(i) / max_x, float(j) / max_y, 0.2f);
  
}

int main() {

    // Image
    int image_width = 1200;
    int image_height = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3* fb;
    hipMallocManaged((void**)&fb, fb_size);

    //timer
    clock_t start, stop;
    start = clock();

    //block
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render <<<blocks, threads >>> (fb, image_width, image_height);
   hipGetLastError();
    hipDeviceSynchronize();

    // Render
    FILE* fp = fopen("output.ppm", "wb");
    fprintf(fp, "P3\n%d %d\n255\n", image_width, image_height);
  //  std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = image_height - 1; j >= 0; --j) {
      //  std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; ++i) {
            size_t pixel_index = j  * image_width + i ;

           int ir = int(255.99*fb[pixel_index].x());
            int ig = int(255.99*fb[pixel_index].y());
            int ib = int(255.99*fb[pixel_index].z());

            fprintf(fp, "%d %d %d\n", ir, ig, ib);
        }
    }
    fclose(fp);
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";
    hipFree(fb);
}